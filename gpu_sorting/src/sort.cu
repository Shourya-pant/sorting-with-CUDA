#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "../include/sort.h"

// CUDA error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Bitonic sort kernel for GPU
__global__ void bitonicSortKernel(int *data, int j, int k) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ixj = i ^ j;
    
    // Only threads with ixj > i are used
    if (ixj > i) {
        // Sort in ascending or descending order
        if ((i & k) == 0) {
            // Ascending
            if (data[i] > data[ixj]) {
                int temp = data[i];
                data[i] = data[ixj];
                data[ixj] = temp;
            }
        } else {
            // Descending
            if (data[i] < data[ixj]) {
                int temp = data[i];
                data[i] = data[ixj];
                data[ixj] = temp;
            }
        }
    }
}

// Float version of bitonic sort kernel for GPU
__global__ void bitonicSortKernelFloat(float *data, int j, int k) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ixj = i ^ j;
    
    // Only threads with ixj > i are used
    if (ixj > i) {
        // Sort in ascending or descending order
        if ((i & k) == 0) {
            // Ascending
            if (data[i] > data[ixj]) {
                float temp = data[i];
                data[i] = data[ixj];
                data[ixj] = temp;
            }
        } else {
            // Descending
            if (data[i] < data[ixj]) {
                float temp = data[i];
                data[i] = data[ixj];
                data[ixj] = temp;
            }
        }
    }
}

// Launch bitonic sort on GPU
void gpuBitonicSort(int *d_data, int n, int threadsPerBlock) {
    // Bitonic sort stages
    for (int k = 2; k <= n; k <<= 1) {
        // Bitonic merge steps
        for (int j = k >> 1; j > 0; j >>= 1) {
            // Calculate grid size based on data size and thread block size
            int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
            
            // Launch kernel
            bitonicSortKernel<<<blocksPerGrid, threadsPerBlock>>>(d_data, j, k);
            
            // Check for errors after kernel launch
            CUDA_CHECK(hipGetLastError());
            
            // Synchronize to ensure kernel completion
            CUDA_CHECK(hipDeviceSynchronize());
        }
    }
}

// Float version of bitonic sort - implementation for the bridge application
extern "C" void bitonicSort(float *d_input, float *d_output, size_t n) {
    // Copy input to output for in-place sorting
    CUDA_CHECK(hipMemcpy(d_output, d_input, n * sizeof(float), hipMemcpyDeviceToDevice));
    
    int threadsPerBlock = 256;
    
    // Bitonic sort stages
    for (int k = 2; k <= n; k <<= 1) {
        // Bitonic merge steps
        for (int j = k >> 1; j > 0; j >>= 1) {
            // Calculate grid size based on data size and thread block size
            int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
            
            // Launch kernel
            bitonicSortKernelFloat<<<blocksPerGrid, threadsPerBlock>>>(d_output, j, k);
            
            // Check for errors after kernel launch
            CUDA_CHECK(hipGetLastError());
            
            // Synchronize to ensure kernel completion
            CUDA_CHECK(hipDeviceSynchronize());
        }
    }
}

// Verify the sorting results
bool verifySorting(const int *data, int n) {
    for (int i = 0; i < n - 1; i++) {
        if (data[i] > data[i + 1]) {
            printf("Sorting verification failed at index %d: %d > %d\n", 
                   i, data[i], data[i + 1]);
            return false;
        }
    }
    return true;
}

// Print array for debugging
void printArray(const int *arr, int n, const char *label) {
    printf("%s: ", label);
    for (int i = 0; i < (n < 20 ? n : 20); i++) {
        printf("%d ", arr[i]);
    }
    if (n > 20) printf("...");
    printf("\n");
}

// Main function to demonstrate GPU bitonic sort
int runGpuSort(int *data, int n) {
    int *d_data = NULL;
    int size = n * sizeof(int);
    
    // Allocate device memory
    CUDA_CHECK(hipMalloc((void**)&d_data, size));
    
    // Copy input data from host to device
    CUDA_CHECK(hipMemcpy(d_data, data, size, hipMemcpyHostToDevice));
    
    // Get device properties
    hipDeviceProp_t deviceProp;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProp, 0));
    
    // Use maximum threads per block (up to 512, which is often optimal)
    int threadsPerBlock = 256;
    if (deviceProp.maxThreadsPerBlock < threadsPerBlock) {
        threadsPerBlock = deviceProp.maxThreadsPerBlock;
    }
    
    // Start timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start));
    
    // Perform the sort
    gpuBitonicSort(d_data, n, threadsPerBlock);
    
    // Stop timing
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    
    // Copy result back to host
    CUDA_CHECK(hipMemcpy(data, d_data, size, hipMemcpyDeviceToHost));
    
    // Clean up
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipFree(d_data));
    
    return milliseconds;
} 